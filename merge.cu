#include "hip/hip_runtime.h"
%%writefile sort1.cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAX_STRING_SIZE 8 // 7 caracteres + terminador nulo

__device__ int compare(const char *str1, const char *str2, int size) {
    for (int i = 0; i < size; ++i)
    {
        char c1 = str1[i];
        char c2 = str2[i];

        if (c1 < c2)
            return -1;
        else if (c1 > c2)
            return 1;
    }
    return 0;
}

__device__ void merge(char *data, char *temp, int left, int mid, int right, int size) {
    int i = left;
    int j = mid + 1;
    int k = left;

    while (i <= mid && j <= right) {
        if (compare(data + i * size, data + j * size, size) <= 0) {
            memcpy(temp + k * size, data + i * size, size);
            i++;
        }
        else {
            memcpy(temp + k * size, data + j * size, size);
            j++;
        }
        k++;
    }

    while (i <= mid) {
        memcpy(temp + k * size, data + i * size, size);
        i++;
        k++;
    }

    while (j <= right) {
        memcpy(temp + k * size, data + j * size, size);
        j++;
        k++;
    }

    //for (int x = left; x <= right; x++)
    //    memcpy(data + x * size, temp + x * size, size);
}

__global__ void mergeKernel(char* arr, char* aux, int tamAtual, int num_items, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int low = idx * width;

    // evitar índices inválidos
    if(low >= num_items - tamAtual || low < 0)
      return;

    int mid = low + tamAtual - 1;
    int high = min(low + width - 1, num_items - 1); // evitar que o high seja maior que o limite superior do vetor

    merge(arr, aux, low, mid, high, MAX_STRING_SIZE);
}

int main(void) {
    int num_items = 0;
    char *h_data = NULL;
    char *d_data = NULL;
    char *auxArr = NULL;

    FILE *file = fopen("quicksort.in", "r");
    if (!file) {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    fscanf(file, "%d", &num_items);

    h_data = (char *)malloc(num_items * MAX_STRING_SIZE);
    if (!h_data) {
        perror("Memory allocation error");
        fclose(file);
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < num_items; i++)
        fscanf(file, "%s", h_data + i * MAX_STRING_SIZE);
    fclose(file);

    hipMalloc((void **)&d_data, num_items * MAX_STRING_SIZE);
    hipMalloc((void **)&auxArr, num_items * MAX_STRING_SIZE);
    hipMemcpy(d_data, h_data, num_items * MAX_STRING_SIZE, hipMemcpyHostToDevice);

    for(int tamAtual = 1; tamAtual < num_items; tamAtual *= 2) {
      int width = tamAtual*2;
      int numSorts = (num_items + width - 1)/width;

      int threadsPerBlock = 64;
      if(numSorts < 32) {
        threadsPerBlock = 2;
      }
      int blocksPerGrid = (numSorts + threadsPerBlock - 1) / threadsPerBlock;
      //printf("numSorts: %d, blocos: %d\n", numSorts, blocksPerGrid);

      //hipMemcpy(auxArr, d_data, num_items * MAX_STRING_SIZE, hipMemcpyHostToDevice);
      mergeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, auxArr, tamAtual, num_items, width);

      //troca de ponteiros
      char* tmp = d_data;
      d_data = auxArr;
      auxArr = tmp;

      //hipDeviceSynchronize();
    }

    char *results_h = (char *)malloc(num_items * MAX_STRING_SIZE);
    hipMemcpy(results_h, d_data, num_items * MAX_STRING_SIZE, hipMemcpyDeviceToHost);

    FILE *out_file = fopen("quicksort.out", "w");
    if (!out_file) {
        perror("Error opening output file");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < num_items; i++)
        fprintf(out_file, "%s\n", results_h + i * MAX_STRING_SIZE);

    fclose(out_file);

    free(h_data);
    free(results_h);
    hipFree(d_data);
    hipFree(auxArr);
    hipDeviceReset();
    exit(EXIT_SUCCESS);
}